
#include <hip/hip_runtime.h>
#include<cstdio>
#include<algorithm>
#include<climits>

extern "C" {

__global__
void initialize(int const *capacity, int *flow, int *excess, int *height, int const s_x, int const s_y, int const N) {
    
    // count coords
    int const x = (blockIdx.x * 32) + threadIdx.x;
    int const y = (blockIdx.y * 32) + threadIdx.y;
    
    if (x>=N || y>=N) return;
    
	int u = y*N + x;
    if (x == s_x && y == s_y) {
        height[u] = N;  // h(s) <- |V|
		excess[u] = INT_MAX/4;
		//printf("START HEIGHT SET TO %d %d\n", height[y*N+x], excess[y*N+x]);
    }
    else {
        height[u] = 0;  // h(u) <- 0
		excess[u] = 0;	
        //Dla dowolnego wierzchołka (x,y) przepustowości krawędzi prowadzących do jego sąsiadów wynoszą: capacity[4∗(N∗y+x)] (krawędź w górę), [...+1] (krawędź w prawo), [...+2] (krawędź w dół), [...3] (krawędź w lewo).
        
        int edge = 4*u; 
        for (int i = 0; i < 4; ++i) {
            flow[edge] = capacity[edge];    // Cf(u, v) <- Cuv
            edge++;
        }
    }
    
    __syncthreads();
    
    if (x == s_x && y == s_y) {
    
        int const vertex_x[] = {0, 1, 0, -1};
        int const vertex_y[] = {-1, 0, 1, 0};
        int edge = 4*u; 
        
        for (int i = 0; i < 4; ++i) {
            flow[edge] = 0; // Cf(s, u) <- 0
            
            int const tmpx = x+vertex_x[i];
            int const tmpy = y+vertex_y[i];
            
            if(tmpx >= 0 && tmpx < N && tmpy>=0 && tmpy < N) {
                excess[N*tmpy+tmpx] = capacity[edge];	// e(u) = C(s, u)
                flow[4*(N*tmpy+tmpx)+((i+2)%4)] += capacity[edge];// + capacity[4*(N*tmpy+tmpx)+(i+2)%4];
            }
            edge++;
            /*
            (x, y-1)
            (x+1, y)
            (x, y+1)
            (x-1, y)
            */
        }
        
        
    }    
    
    return;
}


__global__ 
void push_relabel(int *excess, int *height, int *flow, int const N, int const s_x, int const s_y, int const t_x, int const t_y) {

    // count coords
    int const x = (blockIdx.x * 32) + threadIdx.x;
    int const y = (blockIdx.y * 32) + threadIdx.y;

  if (x>=N || y>=N) return;
   
    if (x == s_x && y == s_y) return;
    if (x == t_x && y == t_y) return;

if(excess[N*y+x]>0){	
  int u = N*y + x;
	
    int temp_e = excess[u];
    int temp_v_x = -1, temp_v_y = -1;
    int temp_h = INT_MAX/2;
    int temp_v_it = -1;
    
    int const vertex_x[] = {0, 1, 0, -1};
    int const vertex_y[] = {-1, 0, 1, 0};
        
	int edge = 4*u; 
	
    for (int i = 0; i < 4; ++i, ++edge) {
    
		if(flow[edge] <= 0) continue;

        int const tmpx = x+vertex_x[i];
        int const tmpy = y+vertex_y[i];
        if(tmpx < 0 || tmpx >= N || tmpy<0 || tmpy >= N) continue;
        	
        int it = N*tmpy + tmpx;
        int try_h = height[it];
		
		if(temp_v_it == -1 || try_h < height[N*temp_v_y+temp_v_x]) {
            temp_h = try_h;
            temp_v_x = tmpx;
            temp_v_y = tmpy; 
            temp_v_it = i;
        }
    }
  
    if (temp_h < height[u]) {
    
        int d = min(temp_e, flow[4*u+temp_v_it]);
    
        atomicAdd(&flow[4*u+temp_v_it], -d);
        atomicAdd(&flow[4*(N*temp_v_y+temp_v_x)+((temp_v_it+2)%4)], d);
        atomicAdd(&excess[u], -d);
        atomicAdd(&excess[N*temp_v_y+temp_v_x], d);
        
    }
    else {
        height[u] = temp_h+1;
    }
}
     

}

__global__
void check_excess(int * excess, int * place_info, int const N, int const s_x, int const s_y, int const t_x, int const t_y) {

    // count coords
    int const x = (blockIdx.x * 32) + threadIdx.x;
    int const y = (blockIdx.y * 32) + threadIdx.y;
    
    if (x>=N || y>=N) return;
    if (s_x == x && s_y == y) return;
	
	
	if (t_x == x && t_y == y) {
		place_info[1] = excess[y*N+x];
	}
	else if (excess[y*N+x] > 0) {
		atomicAdd(&place_info[0], 1);
	}
	
}

}
